#include "hip/hip_runtime.h"
/* Host-side code to perform counting sort 
 * Author: Naga Kandasamy
 * Date modified: May 27, 2020
 * 
 * Compile as follows: make clean && make
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <limits.h>

#include "counting_sort_kernel.cu"

struct timeval start, stop;

/* Do not change the range value */
#define MIN_VALUE 0 
#define MAX_VALUE 255

/* Uncomment to spit out debug info */
// #define DEBUG

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

extern "C" int counting_sort_gold(int *, int *, int, int);
int rand_int(int, int);
void print_array(int *, int);
void print_min_and_max_in_array(int *, int);
void compute_on_device(int *, int *, int, int);
int check_if_sorted(int *, int);
int compare_results(int *, int *, int);

int main(int argc, char **argv)
{
    if (argc < 2) {
        printf("Usage: %s num-elements\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int num_elements = atoi(argv[1]);
    int range = MAX_VALUE - MIN_VALUE;
    int *input_array, *sorted_array_reference, *sorted_array_d;

    /* Populate input array with random integers between [0, RANGE] */
    printf("Generating input array with %d elements in the range 0 to %d\n", num_elements, range);
    input_array = (int *)malloc(num_elements * sizeof(int));
    if (input_array == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    
    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++)
        input_array[i] = rand_int (MIN_VALUE, MAX_VALUE);

#ifdef DEBUG
    print_array(input_array, num_elements);
    print_min_and_max_in_array(input_array, num_elements);
#endif

    /* Sort elements in input array using reference implementation. 
     * The result is placed in sorted_array_reference. */
    printf("\nSorting array on CPU\n");
    gettimeofday(&start, NULL);
    
    int status;
    sorted_array_reference = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_reference == NULL) {
        perror("malloc"); 
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_reference, 0, num_elements);
    
    status = counting_sort_gold(input_array, sorted_array_reference, num_elements, range);
    gettimeofday(&stop, NULL);
    fprintf(stderr, "Gold Execution time = %fs\n", (float) (stop.tv_sec - start.tv_sec\
                    + (stop.tv_usec - start.tv_usec)/(float)1000000));

    if (status == -1) {
        exit(EXIT_FAILURE);
    }

    status = check_if_sorted(sorted_array_reference, num_elements);
    if (status == -1) {
        printf("Error sorting the input array using the reference code\n");
        exit(EXIT_FAILURE);
    }

    printf("Counting sort was successful on the CPU\n");

#ifdef DEBUG
    print_array(sorted_array_reference, num_elements);
#endif

    /* FIXME: Write function to sort elements in the array in parallel fashion. 
     * The result should be placed in sorted_array_mt. */
    printf("\nSorting array on GPU\n");
    sorted_array_d = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_d == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_d, 0, num_elements);

    gettimeofday(&start, NULL);
    compute_on_device(input_array, sorted_array_d, num_elements, range);
    gettimeofday(&stop, NULL);
    fprintf(stderr, "GPU Execution time = %fs\n", (float) (stop.tv_sec - start.tv_sec\
                + (stop.tv_usec - start.tv_usec)/(float)1000000));

#ifdef DEBUG
    print_array(sorted_array_d, num_elements);
#endif

    /* Check the two results for correctness */
    printf("\nComparing CPU and GPU results\n");
    status = compare_results(sorted_array_reference, sorted_array_d, num_elements);
    if (status == 0)
        printf("Test passed\n");
    else
        printf("Test failed\n");

    exit(EXIT_SUCCESS);
}


/* FIXME: Write the GPU implementation of counting sort */
void compute_on_device(int *input_array, int *sorted_array, int num_elements, int range)
{
    int *d_input_array;
    int size = num_elements * sizeof(int);
    hipMalloc((void **)&d_input_array, size);
    hipMemcpy(d_input_array, input_array, size, hipMemcpyHostToDevice);

    int *d_sorted_array;
    size = num_elements * sizeof(int);
    hipMalloc((void **)&d_sorted_array, size);

    int* d_histogram;
    size = (range + 1) * sizeof(int);
    hipMalloc((void **)&d_histogram, size);
    hipMemset(d_histogram, 0, size);

    int* d_scan;
    size = (range + 1) * sizeof(int);
    hipMalloc((void **)&d_scan, (range + 1) * sizeof(int));
    hipMemset(d_scan, 0, size);

    dim3 threads(range + 1);
    dim3 grid(40, 1);
    histogram_kernel_fast<<<grid, threads, size>>>(d_input_array, d_histogram, num_elements, (range + 1));
    hipDeviceSynchronize();
    grid.x = 1;
    counting_sort_kernel<<<grid, threads, 2 * size>>>(d_input_array, d_sorted_array, d_histogram, d_scan, num_elements, range);
    hipMemcpy(sorted_array, d_sorted_array, num_elements * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input_array);
    hipFree(d_sorted_array);
    hipFree(d_histogram);
    hipFree(d_scan);
    return;
}

/* Check if array is sorted */
int check_if_sorted(int *array, int num_elements)
{
    int status = 0;
    int i;
    for (i = 1; i < num_elements; i++) {
        if (array[i - 1] > array[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Check if the arrays elements are identical */ 
int compare_results(int *array_1, int *array_2, int num_elements)
{
    int status = 0;
    int i;
    for (i = 0; i < num_elements; i++) {
        if (array_1[i] != array_2[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Return random integer between [min, max] */ 
int rand_int(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
    return (int)floorf(min + (max - min) * r);
}

/* Print given array */
void print_array(int *this_array, int num_elements)
{
    printf("Array: ");
    int i;
    for (i = 0; i < num_elements; i++)
        printf("%d ", this_array[i]);
    
    printf("\n");
    return;
}

/* Return min and max values in given array */
void print_min_and_max_in_array(int *this_array, int num_elements)
{
    int i;

    int current_min = INT_MAX;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] < current_min)
            current_min = this_array[i];

    int current_max = INT_MIN;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] > current_max)
            current_max = this_array[i];

    printf("Minimum value in the array = %d\n", current_min);
    printf("Maximum value in the array = %d\n", current_max);
    return;
}


